#include "csr.cuh"

CSR_GPU::CSR_GPU(CSR& csr_cpu) {
    size_t row_index_bytes = csr_cpu.csr_row_indices.size()*sizeof(int);
    size_t column_bytes = csr_cpu.csr_columns.size()*sizeof(int);
    size_t data_bytes = csr_cpu.csr_data.size()*sizeof(double);

    hipMalloc(&d_row_indices, row_index_bytes);
    hipMalloc(&d_columns, column_bytes);
    hipMalloc(&d_data, data_bytes);

    hipMemcpy(d_row_indices, csr_cpu.csr_row_indices.data(), row_index_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_columns, csr_cpu.csr_columns.data(), column_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_data, csr_cpu.csr_data.data(), data_bytes, hipMemcpyHostToDevice);
};