// general kernels:

#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void Update_XY(double *X, double *Y, int* step, int max_steps, double time_step, int ncols, int nrows)
{
    // X represents the position of the oscillator
    // Y represents the momentum of the oscillator

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    double pressure = 1.025 * (double) step[0] / max_steps;

    if (ROW < nrows && COL < ncols)
    {
        Y[ROW * ncols + COL] += time_step * (pressure - 1) * X[ROW * ncols + COL];
        X[ROW * ncols + COL] += time_step * (pressure + 1) * Y[ROW * ncols + COL];
    }
}

__global__ void update_window(double *X, int *current_spins, int *final_spins, bool *bifurcated, bool *prev_bifurcated, int* stability, int convergence_threshold, int ncols, int nrows)
{
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    bool equal;
    bool new_bifurcated;

    if (ROW == 0 && COL < ncols)
    {

        // for loop checks if current spin arrangement is identical to previous spin arrangement
        for (int i = 0; i < nrows; i++)
        {
            double val = X[ncols * i + COL];
            int x_sign = (signbit(val) * -2) + 1;
            
            // val / abs(val);
            sum += x_sign * current_spins[ncols * i + COL];
        }

        equal = (sum == nrows);

        bool not_equal = !equal;
        bool not_bifurcated = !bifurcated[COL];

        // if the oscillator has not bifurcated and has not changed, add 1 to the stability
        
        stability[COL] += equal && not_bifurcated;

        // if it has not bifurcated and has changed, reset the stability to 0; otherwise, do not change it

        stability[COL] = not_equal && not_bifurcated ? 0 : stability[COL];

        // if the stability has reached the convergence threshold, then the oscillator has bifurcated (converged)

        bifurcated[COL] = stability[COL] == convergence_threshold - 1;

        // if the oscillator has just bifurcated, then new_bifurcated is true

        new_bifurcated = bifurcated[COL] ^ prev_bifurcated[COL];

        // moves forward in time by moving the present bifurcation array into the previous bifurcation array

        prev_bifurcated[COL] = bifurcated[COL];

        // if the oscillator has bifurcated, we let the sign of the position of the oscillator become the final spins — should only happen once per oscillator

        if (new_bifurcated) {
            for (int i = 0; i < nrows; i++) {
                double val = X[i * ncols + COL];
                int x_sign = (signbit(val) * -2) + 1;
                final_spins[i * ncols + COL] = x_sign;
            }
        }
    }

    // we update our current spins every time this kernel is run, with the same algorithm as the final spins

    if (ROW < nrows && COL < ncols)
    {
        double val = X[ROW * ncols + COL];
        int x_sign = (signbit(val) * -2) + 1;
        // val / abs(val);
        current_spins[ROW * ncols + COL] = x_sign;
    }
}

// MAXCUT-specific kernels:

__global__ void mat_mult_and_confine(int *a_pointers, int *a_indices, double *a_values, double *X, double *Y, double pressure_slope, double time_step, int* step, int max_steps, double xi0, int ncols, int nrows) {

    // the main algorithm

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double total_interaction = 0;

    // calculation of pressure at the time step

    double pressure;
    // pressure = pressure_slope * time_step * step[0];
    pressure = 1.025 * (double) step[0] / max_steps;
    pressure = pressure < 1.0 ? pressure : 1.0;

    if (ROW < nrows && COL < ncols)
    {
        // enforce inelastic walls, by confining all positions to be within the range [-1, 1], and setting momenta to 0 if outside of boundary

        double val = X[ROW * ncols + COL];
        if (abs(val) > 1) 
        {
            X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
            // val / abs(val);
            Y[ROW * ncols + COL] = 0;
        }

        // sparse matrix multiplication: the sparse matrix encodes the interaction coefficients for each spin with all other spins, then calculates with matrix multiplcation
        else
        {
            int start_pointer = a_pointers[ROW];
            int end_pointer = a_pointers[ROW + 1];
            for (int i = start_pointer; i < end_pointer; i++)
            {
                double x_value = X[COL + ncols * a_indices[i]];
                int activated = (signbit(x_value) * -2) + 1;
                // (x_value / abs(x_value));
                total_interaction -= a_values[i] * activated;
            }

            // based on interactions, update the positions and momenta.
            Y[ROW * ncols + COL] += time_step * xi0 * total_interaction;
        }
        // 1: yyx
        // 2: y : yx
        // 3. yxy
        // 4. y : xy

        // update the step number once each time the kernel is run

        if (ROW == 0 && COL == 0) step[0] += 1;
    }
    
}
__global__ void confine(double *X, double *Y, int ncols, int nrows)
{

    // the main algorithm

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    if (ROW < nrows && COL < ncols)
    {
        double val = X[ROW * ncols + COL];
        if (abs(val) > 1) {
                X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
                // val / abs(val);
                Y[ROW * ncols + COL] = 0;
        }
    } 
}

__global__ void step_forward(int* step)
{

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    if (ROW == 0 && COL == 0) step[0] += 1;
}

__global__ void mat_mult(int *a_pointers, int *a_indices, double *a_values, double *X, double *Y, double pressure_slope, double time_step, int* step, int max_steps, double xi0, int ncols, int nrows)
{

    // the main algorithm

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    double total_interaction = 0;

    // calculation of pressure at the time step

    double pressure;
    // pressure = pressure_slope * time_step * step[0];
    pressure = 1.025 * (double) step[0] / max_steps;
    pressure = pressure < 1.0 ? pressure : 1.0;

    if (ROW < nrows && COL < ncols)
    {
        int start_pointer = a_pointers[ROW];
        int end_pointer = a_pointers[ROW + 1];
        for (int i = start_pointer; i < end_pointer; i++)
        {
            double x_value = X[COL + ncols * a_indices[i]];
            int activated = (signbit(x_value) * -2) + 1;
            // (x_value / abs(x_value));
            total_interaction -= a_values[i] * activated;
        }

            // based on interactions, update the positions and momenta.
            Y[ROW * ncols + COL] += time_step * xi0 * total_interaction;
        }
}

__global__ void symplectic_kernel_maxcut(int *a_pointers, int *a_indices, double *a_values, double *X, double *Y, double pressure_slope, double time_step, int* step, int max_steps, double xi0, int ncols, int nrows)
{

    // the main algorithm

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double total_interaction = 0;

    // calculation of pressure at the time step

    double pressure;
    // pressure = pressure_slope * time_step * step[0];
    pressure = 1.025 * (double) step[0] / max_steps;
    pressure = pressure < 1.0 ? pressure : 1.0;

    if (ROW < nrows && COL < ncols)
    {
        // update the step number once each time the kernel is run

        if (ROW == 0 && COL == 0) step[0] += 1;

        // enforce inelastic walls, by confining all positions to be within the range [-1, 1], and setting momenta to 0 if outside of boundary

        double val = X[ROW * ncols + COL];
        if (abs(val) > 1) 
        {
            X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
            // val / abs(val);
            Y[ROW * ncols + COL] = 0;
        }

        // sparse matrix multiplication: the sparse matrix encodes the interaction coefficients for each spin with all other spins, then calculates with matrix multiplcation
        else
        {
            int start_pointer = a_pointers[ROW];
            int end_pointer = a_pointers[ROW + 1];
            for (int i = start_pointer; i < end_pointer; i++)
            {
                double x_value = X[COL + ncols * a_indices[i]];
                int activated = (signbit(x_value) * -2) + 1;
                // (x_value / abs(x_value));
                total_interaction -= a_values[i] * activated;
            }

            // based on interactions, update the positions and momenta.
            Y[ROW * ncols + COL] += time_step * xi0 * total_interaction;
        }

        X[ROW * ncols + COL] += time_step * (pressure + 1) * Y[ROW * ncols + COL];
        Y[ROW * ncols + COL] += time_step * (pressure - 1) * X[ROW * ncols + COL];
        // 1: yyx
        // 2: y : yx
        // 3. yxy
        // 4. y : xy
    }
    
}


__global__ void symplectic_kernel_maxcut_shared_memory(int *a_pointers, int *a_indices, double *a_values, double *X, double *Y, double pressure_slope, double time_step, int* step, int max_steps, double xi0, int ncols, int nrows)
{

    // the main algorithm

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + gridDim.y * blockIdx.z;

    if (ROW >= nrows) return;

    bool confined = false;
    double total_interaction = 0;

    // calculation of pressure at the time step

    double pressure;
    pressure = 1.025 * (double) step[0] / max_steps;
    pressure = pressure < 1.0 ? pressure : 1.0;

    // update the step number once each time the kernel is run

    if (ROW == 0 && COL == 0) step[0] += 1;

    // enforce inelastic walls, by confining all positions to be within the range [-1, 1], and setting momenta to 0 if outside of boundary

    if (COL < ncols) {
        double val = X[ROW * ncols + COL];
        if (abs(val) > 1)
        {
            X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
            // val / abs(val);
            Y[ROW * ncols + COL] = 0;
            confined = true;
        }
    }

    __shared__ float s_values[1024];
    __shared__ int s_indices[1024];

    int start_pointer = a_pointers[ROW];
    int end_pointer = a_pointers[ROW + 1];

    int number_nonzeros_in_row = end_pointer - start_pointer;

    // sparse matrix multiplication: the sparse matrix encodes the interaction coefficients for each spin with all other spins, then calculates with matrix multiplcation

    for (int j = 0; j < ceil((float)number_nonzeros_in_row / blockDim.x); j++)  {
        
        int idx = threadIdx.x + (blockDim.x * j);
        
        if (idx < number_nonzeros_in_row) {
            s_values[threadIdx.x] = a_values[start_pointer + idx];
            s_indices[threadIdx.x] = a_indices[start_pointer + idx];
        }
        else {
            s_values[threadIdx.x] = 0;
            s_indices[threadIdx.x] = -1; // Invalid index
        }

        __syncthreads();

        int end_point = min(blockDim.x, number_nonzeros_in_row - (blockDim.x * j));
        if (COL < ncols) {
            for (int i = 0; i < end_point; i++)  {
                if (s_indices[i] != -1) {
                    float x_value = X[COL + ncols * s_indices[i]];
                    int activated = (signbit(x_value) * -2) + 1;
                    total_interaction -= s_values[i] * activated;
                } else {
                    break;
                }
            }
        }
        __syncthreads();
    }
    
    if (COL < ncols && !confined) {
        int index = COL + ROW * ncols;

        // based on interactions, update the positions and momenta.
        Y[index] += time_step * xi0 * total_interaction;


        X[index] += time_step * (pressure + 1) * Y[index];
        Y[index] += time_step * (pressure - 1) * X[index];
    }
}

__global__ void compute_max_cut(int *a_pointers, int *a_indices, double *a_values, int *final_spins, double *cut_array, int ncols, int nrows)
{
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double tmpSum = 0;

    // calculates the energy of the Hamiltonian according to the following formula:
    // J = interaction matrix, x = spin arrangement
    // 0.5 * x * J * t(x) (all matrix multiplication)

    if (ROW < nrows && COL < ncols)
    {
        int start_pointer = a_pointers[ROW];
        int end_pointer = a_pointers[ROW + 1];
        for (int i = start_pointer; i < end_pointer; i++)
        {
            tmpSum += a_values[i] * final_spins[a_indices[i] * ncols + COL];
        }
        cut_array[ROW * ncols + COL] = tmpSum * final_spins[ROW * ncols + COL] * (0.5);
    }
}

__global__ void sum_max_cut(double *cut_array, double *final_cut_array, double sum, int ncols, int nrows) {
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double tmpSum = 0;

    // calculates the actual maximum cut according to the following formula:
    // H = Hamiltonian energy, S = sum of all weights in MAXCUT graph
    // -0.5 * (H - S)

    if (ROW == 0 && COL < ncols) {
        for (int i = 0; i < nrows; i++) {
            tmpSum += cut_array[i*ncols + COL];
        }
        final_cut_array[COL] = (-0.5) * (tmpSum - sum);
        // final_cut_array[COL] = tmpSum;
    }
}

// Traveling Salesman Problem (TSP)-specific kernels

__global__ void TSP_distances_GEO(double* distances, double* latitudes, double* longitudes, int ncities)
{
    int city1 = blockIdx.x * blockDim.x + threadIdx.x;
    int city2 = blockIdx.y * blockDim.y + threadIdx.y;

    // calculates the distances between all unique pairs of cities according to an adaptation of the Haversine formula
    // the formula used here is from the TSPLIB documentation for "GEO" type distances

    if (city2 < ncities && city1 < city2)
    {
        double R = 6378.388;
        double q1 = cos(longitudes[city1] - longitudes[city2]);
        double q2 = cos(latitudes[city1] - latitudes[city2]);
        double q3 = cos(latitudes[city1] + latitudes[city2]);

        double d = ( R * acos( 0.5 * (( 1.0 + q1 ) * q2 - (1.0 - q1) * q3 )) + 1.0);
        int position = city2 + (city1 * ncities) - ((city1+2)*(city1+1)/ 2);
        distances[position] = (int) d;
    }
}

__global__ void TSP_distances_EUC_2D(double* distances, double* latitudes, double* longitudes, int ncities)
{
    int city1 = blockIdx.x * blockDim.x + threadIdx.x;
    int city2 = blockIdx.y * blockDim.y + threadIdx.y;

    // calculates the distances between all unique pairs of cities according to the euclidean formula
    // the formula used here is from the TSPLIB documentation for "EUC_2D" type distances

    if (city2 < ncities && city1 < city2)
    {
        double xd = latitudes[city1] - latitudes[city2];
        double yd = longitudes[city1] - longitudes[city2];
        double d = pow(xd*xd + yd*yd, 0.5);

        int position = city2 + (city1 * ncities) - ((city1+2)*(city1+1)/ 2);
        distances[position] = (int) d;
    }
}

__global__ void tsp_spin_average(int *spin_array, double* average_array, int ncities, int ncols, int nrows) {
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    // a debugging kernel to determine what the average value for a 
    // particular spin is across the number of agents being run for a TSP graph

    double sum = 0;

    if (ROW < nrows && COL < 1) {
        for (int i = 0; i < ncols; i++) {
            sum += spin_array[ROW * ncols + i];
        }
        average_array[ROW] = sum / ncols;
    }
}

__global__ void update_window_tsp(double *X, int *current_spins, int *final_spins, bool *bifurcated, bool *prev_bifurcated, int* stability, int* valid_list, int convergence_threshold, int* city_visits, int* order_visits, int* step, int ncities, int ncols, int nrows)
{
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    // identical to the update_window kernel above, with small differences that will be pointed out

    bool valid = 1;
    int sum = 0;
    bool equal;
    // bool new_bifurcated;

    if (ROW == 0 && COL < ncols)
    {
        for (int i = 0; i < nrows; i++)
        {
            double val = X[ncols * i + COL];
            int x_sign = (signbit(val) * -2) + 1;
            
            // val / abs(val);
            sum += x_sign * current_spins[ncols * i + COL];

            // the following if statement notes the row and column of every single spin with value "1"

            if (x_sign == 1) {
                int cur_col = i % ncities;
                int cur_row = (i - cur_col) / ncities;
                city_visits[cur_col * ncols + COL] += 1;
                order_visits[cur_row * ncols + COL] += 1;
            }
        }

        // if there is more than one "1" per row or column, for any row or column, then the current spin arrangement is not valid

        for (int i = 0; i < ncities; i++) {
            if (city_visits[i*ncols + COL] != 1 || order_visits[i*ncols + COL] != 1) {
                valid = 0;
                break;
            }
        }

        // reset all values used to 0 for the next iteration of the kernel

        for (int i = 0; i < ncities; i++) {
            city_visits[i*ncols + COL] = 0;
            order_visits[i*ncols + COL] = 0;
        }

        equal = (sum == nrows);

        bool not_equal = !equal;
        bool not_bifurcated = !bifurcated[COL];
        
        stability[COL] += equal && not_bifurcated;
        stability[COL] = not_equal && not_bifurcated ? 0 : stability[COL];
        bifurcated[COL] = stability[COL] == convergence_threshold - 1;

        // new_bifurcated = bifurcated[COL] ^ prev_bifurcated[COL];
        prev_bifurcated[COL] = bifurcated[COL];

        // rather than checking for bifurcation for writing spins to the final spins array, we check for validity, 
        // as this is a stronger and more important metric for TSP graphs

        if (valid) {
            valid_list[COL] = step[0];
            for (int i = 0; i < nrows; i++) {
                double val = X[i*ncols + COL];
                int x_sign = (signbit(val) * -2) + 1;
                final_spins[i * ncols + COL] = x_sign;
            }
        }
    }

    if (ROW < nrows && COL < ncols)
    {
        double val = X[ROW * ncols + COL];
        int x_sign = (signbit(val) * -2) + 1;
        // val / abs(val);
        current_spins[ROW * ncols + COL] = x_sign;
    }
}

__global__ void symplectic_kernel_tsp(int *a_pointers, int *a_indices, double *a_values, double *total_distances, double *X, double *Y, double time_step, int* step, int max_steps, double xi0, int n, double A, double B, double C, int ncities, int ncols, int nrows)
{
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;

    double interaction1 = 0;

    // we initialize time dependent parameters a(t) and b(t)
    // the formulas for these parameters were optimized through trial and error

    double a_t = n * 1.04 * (double) step[0] / max_steps;
    double b_t = 0.5 + ((double) 0.5 * pow( (double) step[0] / (double) max_steps, 1.9));

    // following statements ensures that the parameters never exceed their maximums

    a_t = a_t < n ? a_t : n;
    b_t = b_t < 1 ? b_t : 1;
// 
    if (ROW < nrows && COL < ncols)
    {
        if (ROW == 0 && COL == 0) step[0] += 1;
        double val = X[ROW * ncols + COL];

        if (abs(val) > 1)
        {
            X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
            Y[ROW * ncols + COL] = 0;
        }
        else
        {
            // note that we use the actual value of the position of the oscillator instead of its sign
            // this is ballistic Simulated Bifurcation (bSB) and outperforms discrete Simulated Bifurcation (dSB)
            // there is also a 1.005 multiplier that improves solution quality


            int start_pointer = a_pointers[ROW];
            int end_pointer = a_pointers[ROW + 1];
            for (int i = start_pointer; i < end_pointer; i++)
            {
                double x_value = X[a_indices[i] * ncols + COL];
                // int activated = (signbit(x_value) * -2) + 1;
                double activated = x_value * 1.005;
                interaction1 -= a_values[i] * activated;
            }

            // there is a city dependent second interaction that is gradually increased by the b(t) parameter,
            // such that it allows the system to adiabatically evolve freely before being constrained in the end

            int city_index = ROW % ncities;
            double interaction2 = - 1 * (A * total_distances[city_index] + (ncities-2) * (B + C)) / 2;
            Y[ROW * ncols + COL] += time_step * (xi0 * (2 * interaction1 + b_t * interaction2));
            X[ROW * ncols + COL] += time_step * 1 * Y[ROW * ncols + COL];
            Y[ROW * ncols + COL] += time_step * -(1 - a_t) * X[ROW * ncols + COL];

        }
    }
}

__global__ void symplectic_kernel_tsp_shared_memory(int *a_pointers, int *a_indices, double *a_values, double *total_distances, double *X, double *Y, double time_step, int* step, int max_steps, double xi0, int n, double A, double B, double C, int ncities, int ncols, int nrows)
{

    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    int ROW = blockIdx.y * blockDim.y + gridDim.y * blockIdx.z;

    if (ROW >= nrows) return;

    double interaction1 = 0;
    // bool confined = false;

    // we initialize time dependent parameters a(t) and b(t)
    // the formulas for these parameters were optimized through trial and error

    double a_t = n * 1.04 * (double) step[0] / max_steps;
    double b_t = 0.5 + ((double) 0.5 * pow( (double) step[0] / (double) max_steps, 1.9));

    // following statements ensures that the parameters never exceed their maximums

    a_t = a_t < n ? a_t : n;
    b_t = b_t < 1 ? b_t : 1;

    if (ROW == 0 && COL == 0) step[0] += 1;
    if (COL < ncols) {
        double val = X[ROW * ncols + COL];

        if (abs(val) > 1)
        {
            X[ROW * ncols + COL] = (signbit(val) * -2) + 1;
            Y[ROW * ncols + COL] = 0;
            // confined = true;
        }
    }

    __shared__ float s_values[1024];
    __shared__ int s_indices[1024];

    int start_pointer = a_pointers[ROW];
    int end_pointer = a_pointers[ROW + 1];

    int number_nonzeros_in_row = end_pointer - start_pointer;


    for (int j = 0; j < ceil((float)number_nonzeros_in_row / blockDim.x); j++)  {
        
        int idx = threadIdx.x + (blockDim.x * j);
        
        if (idx < number_nonzeros_in_row) {
            s_values[threadIdx.x] = a_values[start_pointer + idx];
            s_indices[threadIdx.x] = a_indices[start_pointer + idx];
        }
        else {
            s_values[threadIdx.x] = 0;
            s_indices[threadIdx.x] = -1; // Invalid index
        }

        __syncthreads();

        // note that we use the actual value of the position of the oscillator instead of its sign
        // this is ballistic Simulated Bifurcation (bSB) and outperforms discrete Simulated Bifurcation (dSB)
        // there is also a 1.005 multiplier that improves solution quality

        int end_point = min(blockDim.x, number_nonzeros_in_row - (blockDim.x * j));
        if (COL < ncols) {
            for (int i = 0; i < end_point; i++)  {
                if (s_indices[i] != -1) {
                    float x_value = X[COL + ncols * s_indices[i]];
                    interaction1 -= s_values[i] * x_value * 1.005;
                } else {
                    break;
                }
            }
        }
        __syncthreads();
    }

    // there is a city dependent second interaction that is gradually increased by the b(t) parameter,
    // such that it allows the system to adiabatically evolve freely before being constrained in the end

    if (COL < ncols) {
        int city_index = ROW % ncities;
        double interaction2 = - 1 * (A * total_distances[city_index] + (ncities-2) * (B + C)) / 2;
        Y[ROW * ncols + COL] += time_step * (xi0 * (2 * interaction1 + b_t * interaction2));
        X[ROW * ncols + COL] += time_step * 1 * Y[ROW * ncols + COL];
        Y[ROW * ncols + COL] += time_step * -(1 - a_t) * X[ROW * ncols + COL];
    }
}


